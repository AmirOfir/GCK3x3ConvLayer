#include "hip/hip_runtime.h"
#include "FastConv_Conv_Method.cuh"

#include <THC/THCNumerics.cuh>
#include <THC/THCReduceApplyUtils.cuh>
#include <THCUNN/SharedMem.cuh>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>
#include <iostream>


const int WARP_SIZE = 32;
// Crude benchmarks suggest 256 is better than 512 and 1024
// TODO: Autotune/use better heuristics, improve speed more.
const int MAX_BLOCK_SIZE = 256;
const int CUDA_NUM_THREADS = 1024;
const int TOTAL_SHARED_MEMORY_PER_BLOCK_BYTES = 49152;
const int TOTAL_SHARED_MEMORY_PER_BLOCK_FLOATS = TOTAL_SHARED_MEMORY_PER_BLOCK_BYTES / 4;
const int TOTAL_CELLS_POSSIBLE_PER_BLOCK = TOTAL_SHARED_MEMORY_PER_BLOCK_FLOATS / 3;
const int CONSTRAINT_MAX_ROWS_PER_BLOCK = std::min(TOTAL_CELLS_POSSIBLE_PER_BLOCK, CUDA_NUM_THREADS);

using namespace std;
void ThreadsAndBlocksRequiredForInput(int input_dim, int &rows_per_block, int &blocks)
{
    // One thread per input pixel.
    // thread X is the column, Y is the row. 
    // Therefore:
    // Blocks will have A rows, and each block will have 2 overlapping rows with its parent (except the first).
    // Also, block have shared memory (between its threads) that should cover that part of the input, and thec rowwise-results (in-place, i.e. 3*input size)
    int constraintByMaxThreadsPerBlock = CUDA_NUM_THREADS / input_dim;
    int constraintByMaxCellsInSharedMem = TOTAL_CELLS_POSSIBLE_PER_BLOCK / input_dim;

    rows_per_block = std::min(std::min(constraintByMaxCellsInSharedMem, constraintByMaxThreadsPerBlock), input_dim);
    blocks = input_dim / (rows_per_block-2);
}


int GET_BLOCKS(const int N) {
    return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}
static int getGradParamsNumThreads(int batchSize){
    //warp per item in a batch, up to a maximum
    return std::min(batchSize * WARP_SIZE, MAX_BLOCK_SIZE);    
}

///////////////////////////////////////////////////////////////////////
// ROW CONVOLUTION FILTER
///////////////////////////////////////////////////////////////////////

template <typename T>
__global__ void ConvolutionRowwise(const T *input, T *rowwiseResults, 
    int batch_ix, int channel_ix, int input_dim, int input_size, int result_dim)
{
    extern __shared__ int s[];

    // Rowwise results
    int row = threadIdx.y * input_dim;
    int col = threadIdx.x;
    
    // Copy to shared memory
    T inputCellValue = input[col + row + (input_dim * (blockIdx.x * (blockDim.x-2)))];
    s[row + col] = inputCellValue;
    s[row + col +  + input_size] = inputCellValue;
    s[row + col +  + input_size + input_size] = inputCellValue;
    __syncthreads();
    return;
    // Compute rowwise-convolution into shared memory
    
    float* res1 = rowwiseResults + (row * input_dim) + col;
    float* res2 = res1 + (input_dim * input_dim);
    float* res3 = res2 + (input_dim * input_dim);
    input = input 
            /* Current batch */ + (batch_ix /* * num_channels */ * input_dim * input_dim) 
            /* Current input channel: */ + (channel_ix * input_dim * input_dim)
            /* Current Row: */ + (blockIdx.x * input_dim);
    float l1 = input[0],
        l2 = input[1],
        l3 = input[2];
    
    /*for (int i = 3; i < input_dim; ++i)
    {
        *res1 = (l1 + l2 + l3);
        ++res1;
        *res2 = (l1 - l2 + l3);
        ++res2;
        *res3 = (l1 + l2 - l3);
        ++res3;
        l1 = l2;
        l2 = l3;
        l3 = input[i];
    }*/
    /*
    *res1 = (l1 + l2 + l3);
    *res2 = (l1 - l2 + l3);
    *res3 = (l1 + l2 - l3);
    */
}

template <typename T>
__global__ void ConvolutionColwise(const T *rowwiseResults, T *colwiseResults, int input_dim, int result_dim)
{
    // blockDim
    // Z tells us which rowwiseResults matrix to work on {0,1,2}
    // X tells us the rowwiseResults matrix top-row
    // Y tells us the rowwiseResults matrix col

    int topCell = (blockIdx.z *input_dim*result_dim) + (blockIdx.x * result_dim) + blockIdx.y;
    T l1 = rowwiseResults[topCell];
    T l2 = rowwiseResults[topCell + result_dim];
    T l3 = rowwiseResults[topCell + result_dim + result_dim];
    
    topCell = (blockIdx.z * result_dim * result_dim * 3) + (blockIdx.x * result_dim) + blockIdx.y;
    colwiseResults[topCell] = l1 + l2 + l3;
    topCell += result_dim * result_dim;
    colwiseResults[topCell] = l1 - l2 + l3;
    topCell += result_dim * result_dim;
    colwiseResults[topCell] = l1 + l2 - l3;
}

void Convolution3x3ToBasis(
    const torch::Tensor &input, /* Input channel*/
    torch::Tensor &rowwiseResults, /* 3 * input_dim * result_dim */
    torch::Tensor &basisResultsTensor, /* 9 * result_dim * result_dim */
    int batch_ix,
    int channel_ix,
    int input_dim,
    int result_dim,
    int input_size)
{
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    // One thread per input pixel.
    // thread X is the column, Y is the row. 
    // Therefore:
    // Blocks will have A rows, and each block will have 2 overlapping rows with its parent (except the first).
    // Also, block have shared memory (between its threads) that should cover that part of the input, and thec rowwise-results (in-place, i.e. 3*input size)
    int rowsPerBlock, numberOfBlocks;
    rowsPerBlock = std::min(CONSTRAINT_MAX_ROWS_PER_BLOCK / input_dim, input_dim);
    numberOfBlocks = input_dim / (rowsPerBlock-2);
    dim3 threads(input_dim, rowsPerBlock);
    dim3 blocks(numberOfBlocks);
    int sharedMemorySize = input_dim * rowsPerBlock * sizeof(float) * 3;
    // cout << input_dim << "," << rowsPerBlock << "," << numberOfBlocks << endl;
    
    AT_DISPATCH_FLOATING_TYPES(input.type(), "Convolution3x3ToBasis", [&] {
        ConvolutionRowwise <<<blocks, threads, sharedMemorySize, stream>>>
        (
            input.data<float>(),
            rowwiseResults.data<float>(),
            batch_ix, 
            channel_ix,
            input_dim,
            input_size,
            result_dim
        );
    
    //ConvolutionColwise <<< grid_cols, block, 0, stream >>>
    //    (
    //        rowwiseResults.data<float>(),
    //       basisResultsTensor.data<float>(),
    //       input_dim,
    //       result_dim
    //    );
    
    });
  
    THCudaCheck(hipGetLastError());
    THCudaCheck(hipStreamSynchronize(stream));

}
