#include "hip/hip_runtime.h"
#include "FastConv_Conv_Method.cuh"

#include <THC/THCNumerics.cuh>
#include <THC/THCReduceApplyUtils.cuh>
#include <THCUNN/SharedMem.cuh>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>


const int WARP_SIZE = 32;
// Crude benchmarks suggest 256 is better than 512 and 1024
// TODO: Autotune/use better heuristics, improve speed more.
const int MAX_BLOCK_SIZE = 256;
const int CUDA_NUM_THREADS = 1024;

int GET_BLOCKS(const int N) {
    return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}
static int getGradParamsNumThreads(int batchSize){
    //warp per item in a batch, up to a maximum
    return std::min(batchSize * WARP_SIZE, MAX_BLOCK_SIZE);    
}

/*
template <typename T>
__global__ void ConvolutionRowwise(const T *input, T *colwiseResults, int batch_ix,
    int channel_ix,
    int input_dim,
    int result_dim )
{
    T a,b,c;
    T* res1 = colwiseResults;
    T* res2 = res1 + (input_dim * result_dim);
    T* res3 = res2 + (input_dim * result_dim);
}
*/
__global__ void ConvolutionRowwise(const float *input, float *rowwiseResults, int batch_ix,
    int channel_ix,
    int input_dim,
    int result_dim )
{
    float* res1 = rowwiseResults + (blockIdx.x * result_dim);
    float* res2 = res1 + (input_dim * result_dim);
    float* res3 = res2 + (input_dim * result_dim);

    input = input + (blockIdx.x * input_dim);
    float l1 = input[0],
        l2 = input[1],
        l3 = input[2];
    for (int i = 3; i < input_dim; ++i)
    {
        *res1 = (l1 + l2 + l3);
        ++res1;
        *res2 = (l1 - l2 + l3);
        ++res2;
        *res3 = (l1 + l2 - l3);
        ++res3;
        l1 = l2;
        l2 = l3;
        l3 = input[i];
    }

    *res1 = (l1 + l2 + l3);
    *res2 = (l1 - l2 + l3);
    *res3 = (l1 + l2 - l3);
}

void Convolution3x3ToBasis(
    const torch::Tensor &input, /* Input channel*/
    torch::Tensor &colwiseResults, /* 3 * input_dim * result_dim */
    torch::Tensor &basisResultsTensor, /* 9 * result_dim * result_dim */
    int batch_ix,
    int channel_ix,
    int input_dim,
    int result_dim)
{
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    // One thread per output value
    int nthreads = input_dim;
    int blocks = GET_BLOCKS(nthreads);
    dim3 grid(blocks);
    dim3 block(CUDA_NUM_THREADS);
  
  AT_DISPATCH_FLOATING_TYPES(input.type(), "Convolution3x3ToBasis", [&] {
    ConvolutionRowwise <<<grid, block, 0, stream>>>
        (
            input.data<float>(),
            colwiseResults.data<float>(),
            batch_ix, 
            channel_ix,
            input_dim,
            result_dim
        );
  });

  THCudaCheck(hipGetLastError());
}
